/*
 *
 * Layer.cpp
 * Function Defintions for Layer.h.
 *
 */

#include <cmath>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include "Layer.h"
#include "wrapper.h"

Layer::Layer(int input_dim, int output_dim, Activation_Function* activation_function)
	:input_dim(input_dim), output_dim(output_dim), _activation_function(activation_function) {
		hipMalloc((void**)&_weights, sizeof(double) * input_dim * output_dim);
		hipMalloc((void**)&_outputs, sizeof(double) * output_dim);
		hipMalloc((void**)&_error_term, sizeof(double) * output_dim);
		hipMalloc((void**)&_intermediate, sizeof(double) * output_dim);
        	hipMalloc((void**)&_bias, sizeof(double) * output_dim);

		double * weights = (double*)malloc(sizeof(double) * input_dim * output_dim);
		double * outputs = (double*)malloc(sizeof(double) * output_dim);
		double * intermediate = (double*)malloc(sizeof(double) * output_dim);
		double * bias = (double*)malloc(sizeof(double) * output_dim);

		for(int i = 0; i < input_dim * output_dim; i++){
			weights[i] = (rand() % 100) / 100.0;
		}
		
		for(int i = 0; i < output_dim; i++){
			outputs[i] = 0.0;
			intermediate[i] = 0.0;
    			bias[i] = 0.0;
		}
		hipMemcpy(_weights, weights, sizeof(double) * input_dim * output_dim, hipMemcpyHostToDevice);
		hipMemcpy(_intermediate, intermediate, sizeof(double) * output_dim, hipMemcpyHostToDevice);
		hipMemcpy(_outputs, outputs, sizeof(double) * output_dim, hipMemcpyHostToDevice);
		hipMemcpy(_bias, bias, sizeof(double) * output_dim, hipMemcpyHostToDevice);
		free(intermediate);
		free(bias);
		free(outputs);
		free(weights);
}
void Layer::forward(double* inputs){
	forward_wrapper(inputs, _bias, output_dim, input_dim, _intermediate, _weights, _outputs, _activation_function);
}
void Layer::backward(double* actual_outputs, double* activations, Cost_Function *f, double learning_rate, bool final_layer){
	backward_wrapper(activations, actual_outputs, _bias, f, learning_rate, output_dim, input_dim, _intermediate, _weights, _outputs, _activation_function, final_layer, _error_term);
}
void Layer::info(){
    double * weights = (double*)malloc(sizeof(double) * input_dim * output_dim);
    double * outputs = (double*)malloc(sizeof(double) * output_dim);
    double * bias = (double*)malloc(sizeof(double) * output_dim);
    hipMemcpy(weights, _weights, sizeof(double) * output_dim * input_dim, hipMemcpyDeviceToHost);
    hipMemcpy(outputs, _outputs, sizeof(double) * output_dim, hipMemcpyDeviceToHost);
    hipMemcpy(bias, _bias, sizeof(double) * output_dim, hipMemcpyDeviceToHost);
    std::cout << "-------" << std::endl;
	std::cout << "Weights" << std::endl;
	for(int row = 0; row < output_dim; row++){
		for(int col = 0; col < input_dim; col++){
			int index = row * input_dim + col;
			std::cout << weights[index] << "\t";
		}
		std::cout << std::endl;
	}
    
    	std::cout << "Bias" << std::endl;
	for(int i = 0; i < output_dim; i++){
		std::cout << bias[i] << std::endl;
	}
    
	std::cout << "Outputs" << std::endl;
	for(int i = 0; i < output_dim; i++){
		std::cout << outputs[i] << std::endl;
	}
	free(weights);
	free(outputs);
	free(bias);
}

double *Layer::getOutput() {
    double * outputs = (double *) malloc(output_dim * sizeof(double));
    hipMemcpy(outputs, _outputs, sizeof(double) * output_dim, hipMemcpyDeviceToHost);
    return outputs ;
}
double *Layer::getKernelOutput(){
	return _outputs;
}
double *Layer::get_error_term() {
    return _error_term;
}
