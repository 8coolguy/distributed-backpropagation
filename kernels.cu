#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <time.h>
#include "Layer.h"
#include "Neural_Network.h"
#include "Cost_Function.h"
#include "Activation_Function.h"
#include "wrapper.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
	std::cout << "GPUassert: " << hipGetErrorString(code) << \
		", " << file << ", " << line << std::endl;
        if (abort)
            exit(code);
    }
}
#define gpuErrorCheck(ans) {gpuAssert((ans), __FILE__, __LINE__);}

//void Layer::forward(double* inputs){
//	for(int row = 0; row < output_dim; row++){
//        _intermediate[row] = _bias[row];
//		for(int col = 0; col < input_dim; col++){
//			int index = row * input_dim + col;
//			_intermediate[row] += _weights[index] * inputs[col];
//		}
//		_outputs[row] = _activation_function->evaluate(_intermediate[row]);
//	}
//}
__global__ void parallel_forward(double * inputs, double * bias, int output_dim, int input_dim, double * intermediate, double * weights, double * outputs, Activation_Function *activation_function)
{
	int row = threadIdx.x + (blockDim.x * blockIdx.x);	
	int col = threadIdx.y + (blockDim.y * blockIdx.y);	
	if(row >= output_dim) return;
	if(col >= input_dim) return;

	int index = row * input_dim + col;
	//printf("Cuda %d %d %f %f %f\n", row, col, intermediate[row], weights[index], inputs[col]);
	atomicAdd((float*) &intermediate[row], (float)(weights[index] * inputs[col]));
	__syncthreads();
	if(col == 0)
		outputs[row] = 1.0 / (1.0 + exp(-1 * intermediate[row]));
}
void forward_wrapper(double * input, double * bias, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function){

	double *d_input, *d_weights, *d_bias, *d_intermediate, *d_output;
	hipMalloc((void**)&d_input, input_dim * sizeof(double));
	hipMalloc((void**)&d_weights, input_dim * output_dim * sizeof(double));
	hipMalloc((void**)&d_bias, output_dim * sizeof(double)); 
	hipMalloc((void**)&d_intermediate, output_dim*sizeof(double));
	gpuErrorCheck(hipMalloc((void**)&d_output, output_dim * sizeof(double)));

	hipMemcpy(d_input, input, sizeof(double) * input_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, sizeof(double) * input_dim * output_dim, hipMemcpyHostToDevice);
	gpuErrorCheck(hipMemcpy(d_intermediate, bias, sizeof(double) * output_dim, hipMemcpyHostToDevice));

	dim3 block_size(32, 32);
    	dim3 grid_size((output_dim - 1)/32 + 1, (input_dim - 1)/32 + 1);
	parallel_forward<<<grid_size, block_size>>>(d_input, d_bias, output_dim, input_dim, d_intermediate, d_weights, d_output, activation_function);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(double)  * output_dim, hipMemcpyDeviceToHost);
	hipMemcpy(intermediate, d_intermediate, sizeof(double) * output_dim, hipMemcpyDeviceToHost);

	// Free device memory
    	hipFree(d_input);
    	hipFree(d_weights);
    	hipFree(d_bias);
    	hipFree(d_intermediate);
    	hipFree(d_output);

}

//void Layer::backward(double* actual_outputs, Cost_Function *f, double learning_rate){
//    double output_derivatives[output_dim];
//    double intermediate_gradient[output_dim];
//    
//    for (int i = 0; i < output_dim; i++) {
//        output_derivatives[i] = f->derivative(actual_outputs[i], _outputs[i]);
//        intermediate_gradient[i] = _activation_function->derivative(_intermediate[i]);
//    }
//
//    for (int row = 0; row < output_dim; row++) {
//        for (int col = 0; col < input_dim; col++) {
//            int index = row * input_dim + col;
//            _weights[index] -= learning_rate * _weights[index] * output_derivatives[row] * intermediate_gradient[row];
//        }
//        _bias[row] -= learning_rate * output_derivatives[row];
//    }
//}
__global__ void parallel_backward(double * activations, double * actual_outputs, double * bias, Cost_Function * f, double learning_rate, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function, bool final_layer, double * error_term)
{
	int row = threadIdx.x + blockDim.x * blockIdx.x;	
	int col = threadIdx.y + blockDim.y * blockIdx.y;	
	if(row >= output_dim) return;
	if(col >= input_dim) return;

	__shared__ double od[32];
	__shared__ double ig[32];
	
	int index = row * input_dim + col;	

	if (col == 0){
		if(final_layer) od[threadIdx.x] = 2 * (output[row] - actual_outputs[row]);
		else od[threadIdx.x] = actual_outputs[row];
	}
	if(col == 1){
		double sigmoid = 1/(1+ exp(-1 * intermediate[row]));
		ig[threadIdx.x] = sigmoid * (1 - sigmoid);
	}
	if(col == 2) {
		for(int i = 0; i < input_dim; i++){
			int w_index = row * input_dim + i;
			double contribution = weights[w_index];
			error_term[row] += contribution;
		}
	}
	__syncthreads();
	weights[index] -= learning_rate * activations[col] * od[threadIdx.x] * ig[threadIdx.x];
	if (col == 0) error_term[row] = od[threadIdx.x] * ig[threadIdx.x] * error_term[row];
	if (col == 1) bias[row] -= learning_rate * od[threadIdx.x] * ig[threadIdx.x];
	
}
void backward_wrapper(double * activations, double * actual_outputs, double * bias, Cost_Function * f, double learning_rate, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function, bool final_layer, double * error_term){

	double *d_activations,*d_actual_outputs, *d_weights, *d_bias, *d_intermediate, *d_output, *d_error_term;

	//inputs
	hipMalloc((void**)&d_activations, input_dim * sizeof(double));
	hipMalloc((void**)&d_intermediate, output_dim*sizeof(double));
	hipMalloc((void**)&d_actual_outputs, output_dim* sizeof(double));
	gpuErrorCheck(hipMalloc((void**)&d_output, output_dim * sizeof(double)));

	//outputs
	hipMalloc((void**)&d_weights, input_dim * output_dim * sizeof(double));
	hipMalloc((void**)&d_bias, output_dim * sizeof(double)); 
	hipMalloc((void**)&d_error_term, output_dim * sizeof(double));
	

	hipMemcpy(d_actual_outputs, actual_outputs, sizeof(double) * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, sizeof(double) * input_dim * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_bias, bias, sizeof(double) * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_activations, activations, input_dim * sizeof(double), hipMemcpyHostToDevice);
	if(final_layer) hipMemcpy(d_output, output, output_dim * sizeof(double), hipMemcpyHostToDevice);
	gpuErrorCheck(hipMemcpy(d_intermediate, intermediate, sizeof(double) * output_dim, hipMemcpyHostToDevice));

	dim3 block_size(32, 32);
    	dim3 grid_size((output_dim - 1)/32 + 1, (input_dim - 1)/32 + 1);
	parallel_backward<<<grid_size, block_size>>>(d_activations, d_actual_outputs, d_bias, f, learning_rate, output_dim, input_dim, d_intermediate, d_weights, d_output, activation_function, final_layer, d_error_term);
	gpuErrorCheck(hipGetLastError());

	hipDeviceSynchronize();
	hipMemcpy(weights, d_weights, input_dim * output_dim * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(bias, d_bias, output_dim * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(error_term, d_error_term, output_dim * sizeof(double), hipMemcpyDeviceToHost);
	gpuErrorCheck(hipGetLastError());

	// Free device memory
    	hipFree(d_activations);
    	hipFree(d_weights);
    	hipFree(d_bias);
    	hipFree(d_intermediate);
    	hipFree(d_output);
    	hipFree(d_actual_outputs);
    	hipFree(d_error_term);
	gpuErrorCheck(hipGetLastError());
}
