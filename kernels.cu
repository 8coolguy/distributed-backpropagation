#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include "Layer.h"
#include "Neural_Network.h"
#include "Cost_Function.h"
#include "Activation_Function.h"
#include "wrapper.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
	std::cout << "GPUassert: " << hipGetErrorString(code) << \
		", " << file << ", " << line << std::endl;
        if (abort)
            exit(code);
    }
}
#define gpuErrorCheck(ans) {gpuAssert((ans), __FILE__, __LINE__);}

//void Layer::forward(double* inputs){
//	for(int row = 0; row < output_dim; row++){
//        _intermediate[row] = _bias[row];
//		for(int col = 0; col < input_dim; col++){
//			int index = row * input_dim + col;
//			_intermediate[row] += _weights[index] * inputs[col];
//		}
//		_outputs[row] = _activation_function->evaluate(_intermediate[row]);
//	}
//}
__global__ void parallel_forward(double * inputs, double * bias, int output_dim, int input_dim, double * intermediate, double * weights, double * outputs, Activation_Function *activation_function)
{
	int row = threadIdx.x + (blockDim.x * blockIdx.x);	
	int col = threadIdx.y + (blockDim.y * blockIdx.y);	
	if(row >= output_dim) return;
	if(col >= input_dim) return;

	int index = row * input_dim + col;
	//printf("Cuda %d %d %f %f %f\n", row, col, intermediate[row], weights[index], inputs[col]);
	atomicAdd((float*) &intermediate[row], (float) (weights[index] * inputs[col]));
	__syncthreads();
	if(col == 0)
		outputs[row] = 1.0 / (1.0 + exp(-1 * intermediate[row]));
}
void forward_wrapper(double * input, double * bias, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function){

	double *d_input, *d_weights, *d_bias, *d_intermediate, *d_output;
	hipMalloc((void**)&d_input, input_dim * sizeof(double));
	hipMalloc((void**)&d_weights, input_dim * output_dim * sizeof(double));
	hipMalloc((void**)&d_bias, input_dim * sizeof(double)); 
	hipMalloc((void**)&d_intermediate, output_dim*sizeof(double));
	hipMalloc((void**)&d_output, output_dim * sizeof(double));

	hipMemcpy(d_input, input, sizeof(double) * input_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, sizeof(double) * input_dim * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_intermediate, bias, sizeof(double) * output_dim, hipMemcpyHostToDevice);

	dim3 block_size(32, 32);
    	dim3 grid_size((output_dim - 1)/32 + 1, (input_dim - 1)/32 + 1);
	parallel_forward<<<grid_size, block_size>>>(d_input, d_bias, output_dim, input_dim, d_intermediate, d_weights, d_output, activation_function);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(double)  * output_dim, hipMemcpyDeviceToHost);
	hipMemcpy(intermediate, d_intermediate, sizeof(double) * output_dim, hipMemcpyDeviceToHost);

	// Free device memory
    	hipFree(d_input);
    	hipFree(d_weights);
    	hipFree(d_bias);
    	hipFree(d_intermediate);
    	hipFree(d_output);

}

//void Layer::backward(double* actual_outputs, Cost_Function *f, double learning_rate){
//    double output_derivatives[output_dim];
//    double intermediate_gradient[output_dim];
//    
//    for (int i = 0; i < output_dim; i++) {
//        output_derivatives[i] = f->derivative(actual_outputs[i], _outputs[i]);
//        intermediate_gradient[i] = _activation_function->derivative(_intermediate[i]);
//    }
//
//    for (int row = 0; row < output_dim; row++) {
//        for (int col = 0; col < input_dim; col++) {
//            int index = row * input_dim + col;
//            _weights[index] -= learning_rate * _weights[index] * output_derivatives[row] * intermediate_gradient[row];
//        }
//        _bias[row] -= learning_rate * output_derivatives[row];
//    }
//}
/*
__global__ void parallel_backward(double * actual_outputs, double * bias, double * output_derivatives, double * intermediate_gradient, Cost_Function * f, int learning_rate, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function)
{
	int row = threadIdx.x + blockDim.x * blockIdx.x;	
	int col = threadIdx.y + blockDim.y * blockIdx.y;	
	int index = row * input_dim + col;	

	if (col == 0) output_derivatives[row] = f->derivative(actual_outputs[row], _outputs[row]);
	if (col == 1) intermediate_gradient[row] = _activation_function->derivative(_intermediate[row]);
	__syncthreads();

	weights[index] -= learning_rate * weights[index] * output_derivatives[row] * intermediate_gradient[row];
	if (col == 0) bias[row] -= learning_rate * output_derivatives[row];
	__syncthreads();
	
}


*/
