#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include "Layer.h"
#include "Neural_Network.h"
#include "Cost_Function.h"
#include "Activation_Function.h"
#include "wrapper.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
	std::cout << "GPUassert: " << hipGetErrorString(code) << \
		", " << file << ", " << line << std::endl;
        if (abort)
            exit(code);
    }
}
#define gpuErrorCheck(ans) {gpuAssert((ans), __FILE__, __LINE__);}

//void Layer::forward(double* inputs){
//	for(int row = 0; row < output_dim; row++){
//        _intermediate[row] = _bias[row];
//		for(int col = 0; col < input_dim; col++){
//			int index = row * input_dim + col;
//			_intermediate[row] += _weights[index] * inputs[col];
//		}
//		_outputs[row] = _activation_function->evaluate(_intermediate[row]);
//	}
//}
__global__ void parallel_forward(double * inputs, double * bias, int output_dim, int input_dim, double * intermediate, double * weights, double * outputs, Activation_Function *activation_function)
{
	int row = threadIdx.x + (blockDim.x * blockIdx.x);	
	int col = threadIdx.y + (blockDim.y * blockIdx.y);	
	if(row >= output_dim) return;
	if(col >= input_dim) return;

	int index = row * input_dim + col;
	//printf("Cuda %d %d %f %f %f\n", row, col, intermediate[row], weights[index], inputs[col]);
	atomicAdd((float*) &intermediate[row], (float) (weights[index] * inputs[col]));
	__syncthreads();
	if(col == 0)
		outputs[row] = 1.0 / (1.0 + exp(-1 * intermediate[row]));
}
void forward_wrapper(double * input, double * bias, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function){

	double *d_input, *d_weights, *d_bias, *d_intermediate, *d_output;
	hipMalloc((void**)&d_input, input_dim * sizeof(double));
	hipMalloc((void**)&d_weights, input_dim * output_dim * sizeof(double));
	hipMalloc((void**)&d_bias, output_dim * sizeof(double)); 
	hipMalloc((void**)&d_intermediate, output_dim*sizeof(double));
	hipMalloc((void**)&d_output, output_dim * sizeof(double));

	hipMemcpy(d_input, input, sizeof(double) * input_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, sizeof(double) * input_dim * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_intermediate, bias, sizeof(double) * output_dim, hipMemcpyHostToDevice);

	dim3 block_size(32, 32);
    	dim3 grid_size((output_dim - 1)/32 + 1, (input_dim - 1)/32 + 1);
	parallel_forward<<<grid_size, block_size>>>(d_input, d_bias, output_dim, input_dim, d_intermediate, d_weights, d_output, activation_function);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(double)  * output_dim, hipMemcpyDeviceToHost);
	hipMemcpy(intermediate, d_intermediate, sizeof(double) * output_dim, hipMemcpyDeviceToHost);

	// Free device memory
    	hipFree(d_input);
    	hipFree(d_weights);
    	hipFree(d_bias);
    	hipFree(d_intermediate);
    	hipFree(d_output);

}

//void Layer::backward(double* actual_outputs, Cost_Function *f, double learning_rate){
//    double output_derivatives[output_dim];
//    double intermediate_gradient[output_dim];
//    
//    for (int i = 0; i < output_dim; i++) {
//        output_derivatives[i] = f->derivative(actual_outputs[i], _outputs[i]);
//        intermediate_gradient[i] = _activation_function->derivative(_intermediate[i]);
//    }
//
//    for (int row = 0; row < output_dim; row++) {
//        for (int col = 0; col < input_dim; col++) {
//            int index = row * input_dim + col;
//            _weights[index] -= learning_rate * _weights[index] * output_derivatives[row] * intermediate_gradient[row];
//        }
//        _bias[row] -= learning_rate * output_derivatives[row];
//    }
//}
__global__ void parallel_backward(double * activations, double * actual_outputs, double * bias, Cost_Function * f, int learning_rate, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function, bool final_layer, double * error_term, double * output_derivatives, double * intermediate_gradient)
{
	int row = threadIdx.x + blockDim.x * blockIdx.x;	
	int col = threadIdx.y + blockDim.y * blockIdx.y;	
	if(row >= output_dim) return;
	if(col >= input_dim) return;
	
	int index = row * input_dim + col;	

	if (col == 0){
		if(final_layer) output_derivatives[row] = 2 * (output[row] - actual_outputs[row]);
		else output_derivatives[row] = actual_outputs[row];
	}
	if (col == 1){
		double sigmoid = 1/(1+ exp(-1 * intermediate[row]));
		intermediate_gradient[row] = sigmoid * (1 - sigmoid);
	}
	if (col == 2) error_term[row] = 0;
	__syncthreads();

	atomicAdd((float*)&error_term[row], (float)(weights[index])); 

	weights[index] -= learning_rate * activations[col] * output_derivatives[row] * intermediate_gradient[row];

	if (col == 0) error_term[row] = output_derivatives[row] * intermediate_gradient[row] * error_term[row];
	if (col == 1) bias[row] -= learning_rate * output_derivatives[row] * intermediate_gradient[row];
	__syncthreads();
	
}
void backward_wrapper(double * activations, double * actual_outputs, double * bias, Cost_Function * f, int learning_rate, int output_dim, int input_dim, double * intermediate, double * weights, double * output, Activation_Function *activation_function, bool final_layer, double * error_term){

	double *d_activations,*d_actual_outputs, *d_weights, *d_bias, *d_intermediate, *d_output, *d_intermediate_gradient, *d_error_term;
	double *d_od;

	hipMalloc((void**)&d_intermediate_gradient, output_dim * sizeof(double));
	hipMalloc((void**)&d_od, output_dim * sizeof(double));
	//inputs
	hipMalloc((void**)&d_activations, input_dim * sizeof(double));
	hipMalloc((void**)&d_intermediate, output_dim*sizeof(double));
	hipMalloc((void**)&d_actual_outputs, output_dim* sizeof(double));
	hipMalloc((void**)&d_output, output_dim * sizeof(double));

	//outputs
	hipMalloc((void**)&d_weights, input_dim * output_dim * sizeof(double));
	hipMalloc((void**)&d_bias, output_dim * sizeof(double)); 
	hipMalloc((void**)&d_error_term, output_dim * sizeof(double));
	

	hipMemcpy(d_actual_outputs, actual_outputs, sizeof(double) * input_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, sizeof(double) * input_dim * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_bias, bias, sizeof(double) * output_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_activations, activations, input_dim * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_output, output, output_dim * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_intermediate, intermediate, sizeof(double) * output_dim, hipMemcpyHostToDevice);

	dim3 block_size(32, 32);
    	dim3 grid_size((output_dim - 1)/32 + 1, (input_dim - 1)/32 + 1);
	parallel_backward<<<grid_size, block_size>>>
		(d_activations, d_actual_outputs, d_bias, f, learning_rate, output_dim, input_dim, d_intermediate, d_weights, d_output, activation_function, final_layer, d_error_term, d_od, d_intermediate_gradient);

	hipDeviceSynchronize();
	hipMemcpy(weights, d_weights, input_dim * output_dim * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(bias, d_bias, output_dim * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(error_term, d_error_term, output_dim * sizeof(double), hipMemcpyDeviceToHost);

	// Free device memory
    	hipFree(d_activations);
    	hipFree(d_weights);
    	hipFree(d_bias);
    	hipFree(d_intermediate);
    	hipFree(d_output);
    	hipFree(d_intermediate_gradient);
    	hipFree(d_od);
    	hipFree(d_actual_outputs);
    	hipFree(d_error_term);
}
